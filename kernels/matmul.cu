
#include <hip/hip_runtime.h>
#define TileWidth 16

__global__ void matmul_kernel_base(const float *A, const float *B, float *result, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = 0; i< K; i++){
        if(row < M && col < N)
            result[row * N + col] += A[row * K + i] * B[col * K + i];
    }
    
}

void launch_matmul_naive(const float *A, const float *B, float *result, int M, int N, int K){
    dim3 block(TileWidth, TileWidth);
    dim3 grid((M + TileWidth - 1) / TileWidth, (N + TileWidth - 1) / TileWidth);
    matmul_kernel_base<<<grid, block>>>(A, B, result, M, N, K);
    hipDeviceSynchronize();
}



__global__ void matmul_kernel_tile(const float *A, const float *B, float *result, int M, int N, int K) {
    // define shared memory  equals to block shape
    __shared__ float subTiledA[TileWidth][TileWidth];
    __shared__ float subTiledB[TileWidth][TileWidth];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x; 
    float resultValue = 0.f;
    for(int m = 0; m < K / TileWidth; m++){
        // fill data in subTiledA and B
        subTiledA[threadIdx.y][threadIdx.x] = A[row * K + m * TileWidth + threadIdx.x];    //A[row][m * TileWidth + threadIdx.x]; 
        subTiledB[threadIdx.y][threadIdx.x] = B[col * K + m * TileWidth + threadIdx.y]; 
        __syncthreads(); // wait all threads in this block  finish loading data from global memory
        // calc value
        for(int i = 0; i < TileWidth; i++){
            resultValue += subTiledA[threadIdx.y][m] * subTiledB[m][threadIdx.x];
        }
        __syncthreads(); // wait all the m-th subtile cal done
    } 
    result[row * N + col] = resultValue; 
     
    
}

void launch_matmul_tiled(const float *A, const float *B, float *result, int M, int N, int K){
    dim3 block(TileWidth, TileWidth);
    dim3 grid((N + TileWidth - 1) / TileWidth, (M + TileWidth - 1) / TileWidth);
    matmul_kernel_tile<<<grid, block>>>(A, B, result, M, N, K);
    hipDeviceSynchronize();
}