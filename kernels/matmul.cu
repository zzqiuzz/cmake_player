#define TileWidth 32 
#define TileKWidth 32 // 32
#define TileMWidth 32 
#define TileNWidth 32 

#include <hip/hip_runtime.h>
#include <iostream>
using std::cout;
using std::endl;
__global__ void matmul_kernel_base(const float *A, const float *B, float *result, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = 0; i< K; i++){
        if(row < M && col < N)
            result[row * N + col] += A[row * K + i] * B[i * N + col];
    }
    
}

void launch_matmul_naive(const float *A, const float *B, float *result, int M, int N, int K){
    dim3 block(TileWidth, TileWidth);
    dim3 grid((N + TileWidth - 1) / TileWidth, (M + TileWidth - 1) / TileWidth);
    cout << "block.x = " << block.x << " block.y = " << block.y << " block.z = " << block.z << endl;
    cout << "grid.x = " << grid.x << " grid.y= " << grid.y << " grid.z = " << grid.z;
    cout << endl;
    matmul_kernel_base<<<grid, block>>>(A, B, result, M, N, K);
    hipDeviceSynchronize();
}



__global__ void matmul_kernel_tile(const float *A, const float *B, float *result, int M, int N, int K) {
    // define shared memory  equals to block shape
    __shared__ float subTiledA[TileMWidth][TileKWidth]; // 32 x 16
    __shared__ float subTiledB[TileKWidth][TileNWidth]; // 16 x 32

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x; 
    float resultValue = 0.f;
    for(int m = 0; m < (K + TileKWidth - 1) / TileKWidth; m++){
        // fill data in subTiledA and B
        int a_row = row;
        int a_col = m * TileKWidth + threadIdx.x;
        if(threadIdx.x < TileKWidth){
            if(a_row < M && a_col < K)
                subTiledA[threadIdx.y][threadIdx.x] = A[row * K + m * TileKWidth + threadIdx.x];    //A[row][m * TileWidth + threadIdx.x]; 
            else
                subTiledA[threadIdx.y][threadIdx.x] = 0.f;
        }

        int b_row = m * TileKWidth + threadIdx.y;
        int b_col = col;
        if(threadIdx.y < TileKWidth){
            if(b_row < K && b_col < N)
                // subTiledB[threadIdx.y][threadIdx.x] = B[col * K + m * TileKWidth + threadIdx.y];  error
                subTiledB[threadIdx.y][threadIdx.x] = B[b_row * N + b_col]; // 行优先的话，按照行来索引！
            else    
                subTiledB[threadIdx.y][threadIdx.x] = 0.f;
        }
        __syncthreads(); // wait all threads in this block  finish loading data from global memory
        // calc value
        for(int i = 0; i < TileKWidth; i++){
            resultValue += subTiledA[threadIdx.y][i] * subTiledB[i][threadIdx.x];
        }
        __syncthreads(); // wait all the m-th subtile cal done
    } 
    if(row < M && col < N)
        result[row * N + col] = resultValue; 
     
    
}

void launch_matmul_tiled(const float *A, const float *B, float *result, int M, int N, int K){
    dim3 block(TileWidth, TileWidth);
    dim3 grid((N + TileWidth - 1) / TileWidth, (M + TileWidth - 1) / TileWidth); 
    cout << "block.x = " << block.x << " block.y = " << block.y << " block.z = " << block.z << endl;
    cout << "grid.x = " << grid.x << " grid.y= " << grid.y << " grid.z = " << grid.z;
    cout << endl;
    matmul_kernel_tile<<<grid, block>>>(A, B, result, M, N, K);
    hipDeviceSynchronize();
}